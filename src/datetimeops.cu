
#include <gdf/gdf.h>
#include <gdf/utils.h>
#include <gdf/errorutils.h>


#include <hip/hip_runtime.h>
#include <vector>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <thrust/execution_policy.h>
#include <thrust/iterator/iterator_adaptor.h>
#include <thrust/device_vector.h>

/*
 * from http://howardhinnant.github.io/date_algorithms.html
 * int z = unixTime/86400000;
 * z += 719468;
    const Int era = (z >= 0 ? z : z - 146096) / 146097;
    const unsigned doe = static_cast<unsigned>(z - era * 146097);          // [0, 146096]
    const unsigned yoe = (doe - doe/1460 + doe/36524 - doe/146096) / 365;  // [0, 399]
    const Int y = static_cast<Int>(yoe) + era * 400;
    const unsigned doy = doe - (365*yoe + yoe/4 - yoe/100);                // [0, 365]
    const unsigned mp = (5*doy + 2)/153;                                   // [0, 11]
    const unsigned d = doy - (153*mp+2)/5 + 1;                             // [1, 31]
    const unsigned m = mp + (mp < 10 ? 3 : -9);                            // [1, 12]
    return std::tuple<Int, unsigned, unsigned>(y + (m <= 2), m, d);
 */


struct gdf_extract_datetime_year_date64_op : public thrust::unary_function<int64_t, int16_t>
{

	__host__ __device__
	int16_t operator()(int64_t unixTime) // unixTime is milliseconds since the UNIX epoch
	{
		const int z = ((unixTime >= 0 ? unixTime : unixTime - 86399999) / 86400000) + 719468;
		const int era = (z >= 0 ? z : z - 146096) / 146097;
		const unsigned doe = static_cast<unsigned>(z - era * 146097);          // [0, 146096]
		const unsigned yoe = (doe - doe/1460 + doe/36524 - doe/146096) / 365;  // [0, 399]
		const int y = static_cast<int>(yoe) + era * 400;
		const unsigned doy = doe - (365*yoe + yoe/4 - yoe/100);                // [0, 365]
		const unsigned mp = (5*doy + 2)/153;                                   // [0, 11]
//		const unsigned d = doy - (153*mp+2)/5 + 1;                             // [1, 31]
		const unsigned m = mp + (mp < 10 ? 3 : -9);                            // [1, 12]
		if (m <= 2)
			return y + 1;
		else
			return y;
	}
};

struct gdf_extract_datetime_month_date64_op : public thrust::unary_function<int64_t, int16_t>
{

	__host__ __device__
	int16_t operator()(int64_t unixTime) // unixTime is milliseconds since the UNIX epoch
	{
		const int z = ((unixTime >= 0 ? unixTime : unixTime - 86399999) / 86400000) + 719468;
		const int era = (z >= 0 ? z : z - 146096) / 146097;
		const unsigned doe = static_cast<unsigned>(z - era * 146097);          // [0, 146096]
		const unsigned yoe = (doe - doe/1460 + doe/36524 - doe/146096) / 365;  // [0, 399]
//		const int y = static_cast<int>(yoe) + era * 400;
		const unsigned doy = doe - (365*yoe + yoe/4 - yoe/100);                // [0, 365]
		const unsigned mp = (5*doy + 2)/153;                                   // [0, 11]
//		const unsigned d = doy - (153*mp+2)/5 + 1;                             // [1, 31]
		return mp + (mp < 10 ? 3 : -9);                            // [1, 12]

	}
};

struct gdf_extract_datetime_day_date64_op : public thrust::unary_function<int64_t, int16_t>
{

	__host__ __device__
	int16_t operator()(int64_t unixTime) // unixTime is milliseconds since the UNIX epoch
	{
		const int z = ((unixTime >= 0 ? unixTime : unixTime - 86399999) / 86400000) + 719468;
		const int era = (z >= 0 ? z : z - 146096) / 146097;
		const unsigned doe = static_cast<unsigned>(z - era * 146097);          // [0, 146096]
		const unsigned yoe = (doe - doe/1460 + doe/36524 - doe/146096) / 365;  // [0, 399]
//		const int y = static_cast<int>(yoe) + era * 400;
		const unsigned doy = doe - (365*yoe + yoe/4 - yoe/100);                // [0, 365]
		const unsigned mp = (5*doy + 2)/153;                                   // [0, 11]
		return doy - (153*mp+2)/5 + 1;                             // [1, 31]
	}
};


struct gdf_extract_datetime_hour_date64_op : public thrust::unary_function<int64_t, int16_t>
{
	__host__ __device__
	int16_t operator()(int64_t unixTime) // unixTime is milliseconds since the UNIX epoch
	{
		return unixTime >= 0 ? ((unixTime % 86400000)/3600000) : ((86400000+ (unixTime % 86400000))/3600000);
	}
};

struct gdf_extract_datetime_minute_date64_op : public thrust::unary_function<int64_t, int16_t>
{
	__host__ __device__
	int16_t operator()(int64_t unixTime) // unixTime is milliseconds since the UNIX epoch
	{
		return unixTime >= 0 ? ((unixTime % 3600000)/60000) :  ((3600000 + (unixTime % 3600000))/60000);
	}
};

struct gdf_extract_datetime_second_date64_op : public thrust::unary_function<int64_t, int16_t>
{
	__host__ __device__
	int16_t operator()(int64_t unixTime) // unixTime is milliseconds since the UNIX epoch
	{
		return unixTime >= 0 ? ((unixTime % 60000)/1000) : ((60000 + (unixTime % 60000))/1000);
	}
};

struct gdf_extract_datetime_millisecond_date64_op : public thrust::unary_function<int64_t, int16_t>
{
	__host__ __device__
	int16_t operator()(int64_t unixTime) // unixTime is milliseconds since the UNIX epoch
	{
		return unixTime >= 0 ? unixTime % 1000 :  1000 + unixTime % 1000;
	}
};

struct gdf_extract_datetime_year_date32_op : public thrust::unary_function<int32_t, int16_t>
{

	__host__ __device__
	int16_t operator()(int32_t unixDate) // unixDate is days since the UNIX epoch
	{
		const int z = unixDate + 719468;
		const int era = (z >= 0 ? z : z - 146096) / 146097;
		const unsigned doe = static_cast<unsigned>(z - era * 146097);          // [0, 146096]
		const unsigned yoe = (doe - doe/1460 + doe/36524 - doe/146096) / 365;  // [0, 399]
		const int y = static_cast<int>(yoe) + era * 400;
		const unsigned doy = doe - (365*yoe + yoe/4 - yoe/100);                // [0, 365]
		const unsigned mp = (5*doy + 2)/153;                                   // [0, 11]
		//		const unsigned d = doy - (153*mp+2)/5 + 1;                             // [1, 31]
		const unsigned m = mp + (mp < 10 ? 3 : -9);                            // [1, 12]
		if (m <= 2)
			return y + 1;
		else
			return y;
	}
};

struct gdf_extract_datetime_month_date32_op : public thrust::unary_function<int32_t, int16_t>
{

	__host__ __device__
	int16_t operator()(int32_t unixDate) // unixDate is days since the UNIX epoch
	{
		const int z = unixDate + 719468;
		const int era = (z >= 0 ? z : z - 146096) / 146097;
		const unsigned doe = static_cast<unsigned>(z - era * 146097);          // [0, 146096]
		const unsigned yoe = (doe - doe/1460 + doe/36524 - doe/146096) / 365;  // [0, 399]
		//		const int y = static_cast<int>(yoe) + era * 400;
		const unsigned doy = doe - (365*yoe + yoe/4 - yoe/100);                // [0, 365]
		const unsigned mp = (5*doy + 2)/153;                                   // [0, 11]
		//		const unsigned d = doy - (153*mp+2)/5 + 1;                             // [1, 31]
		return mp + (mp < 10 ? 3 : -9);
	}
};

struct gdf_extract_datetime_day_date32_op : public thrust::unary_function<int32_t, int16_t>
{

	__host__ __device__
	int16_t operator()(int32_t unixDate) // unixDate is days since the UNIX epoch
	{
		const int z = unixDate + 719468;
		const int era = (z >= 0 ? z : z - 146096) / 146097;
		const unsigned doe = static_cast<unsigned>(z - era * 146097);          // [0, 146096]
		const unsigned yoe = (doe - doe/1460 + doe/36524 - doe/146096) / 365;  // [0, 399]
		//		const int y = static_cast<int>(yoe) + era * 400;
		const unsigned doy = doe - (365*yoe + yoe/4 - yoe/100);                // [0, 365]
		const unsigned mp = (5*doy + 2)/153;                                   // [0, 11]
		return doy - (153*mp+2)/5 + 1;                             // [1, 31]
	}
};





gdf_error gdf_extract_datetime_year(gdf_column *input, gdf_column *output) {

	GDF_REQUIRE(input->size == output->size, GDF_COLUMN_SIZE_MISMATCH);
	GDF_REQUIRE(output->dtype == GDF_INT16, GDF_UNSUPPORTED_DTYPE);

	hipStream_t stream;
	hipStreamCreate(&stream);

	gdf_size_type num_chars_bitmask = ( ( input->size +( GDF_VALID_BITSIZE - 1)) / GDF_VALID_BITSIZE );
	thrust::copy(thrust::cuda::par.on(stream), input->valid, input->valid + num_chars_bitmask, output->valid); // copy over valid bitmask

	if ( input->dtype == GDF_DATE64 ) {
		thrust::device_ptr<int64_t> input_ptr((int64_t *) input->data);
		thrust::device_ptr<int16_t> output_ptr((int16_t *) output->data);
		gdf_extract_datetime_year_date64_op op;
		thrust::transform(thrust::cuda::par.on(stream), thrust::detail::make_normal_iterator(input_ptr),
				thrust::detail::make_normal_iterator(input_ptr) + input->size, thrust::detail::make_normal_iterator(output_ptr), op);

	}else if (input->dtype == GDF_DATE32) {
		thrust::device_ptr<int32_t> input_ptr((int32_t *) input->data);
		thrust::device_ptr<int16_t> output_ptr((int16_t *) output->data);
		gdf_extract_datetime_year_date32_op op;
		thrust::transform(thrust::cuda::par.on(stream), thrust::detail::make_normal_iterator(input_ptr),
				thrust::detail::make_normal_iterator(input_ptr) + input->size, thrust::detail::make_normal_iterator(output_ptr), op);
	} else {
		return GDF_UNSUPPORTED_DTYPE;
	}

	hipStreamSynchronize(stream);
	hipStreamDestroy(stream);

	return GDF_SUCCESS;
}

gdf_error gdf_extract_datetime_month(gdf_column *input, gdf_column *output) {

	GDF_REQUIRE(input->size == output->size, GDF_COLUMN_SIZE_MISMATCH);
	GDF_REQUIRE(output->dtype == GDF_INT16, GDF_UNSUPPORTED_DTYPE);

	hipStream_t stream;
	hipStreamCreate(&stream);

	gdf_size_type num_chars_bitmask = ( ( input->size +( GDF_VALID_BITSIZE - 1)) / GDF_VALID_BITSIZE );
	thrust::copy(thrust::cuda::par.on(stream), input->valid, input->valid + num_chars_bitmask, output->valid); // copy over valid bitmask

	if ( input->dtype == GDF_DATE64 ) {
		thrust::device_ptr<int64_t> input_ptr((int64_t *) input->data);
		thrust::device_ptr<int16_t> output_ptr((int16_t *) output->data);
		gdf_extract_datetime_month_date64_op op;
		thrust::transform(thrust::cuda::par.on(stream), thrust::detail::make_normal_iterator(input_ptr),
				thrust::detail::make_normal_iterator(input_ptr) + input->size, thrust::detail::make_normal_iterator(output_ptr), op);

	}else if (input->dtype == GDF_DATE32) {
		thrust::device_ptr<int32_t> input_ptr((int32_t *) input->data);
		thrust::device_ptr<int16_t> output_ptr((int16_t *) output->data);
		gdf_extract_datetime_month_date32_op op;
		thrust::transform(thrust::cuda::par.on(stream), thrust::detail::make_normal_iterator(input_ptr),
				thrust::detail::make_normal_iterator(input_ptr) + input->size, thrust::detail::make_normal_iterator(output_ptr), op);
	} else {
		return GDF_UNSUPPORTED_DTYPE;
	}

	hipStreamSynchronize(stream);
	hipStreamDestroy(stream);

	return GDF_SUCCESS;
}

gdf_error gdf_extract_datetime_day(gdf_column *input, gdf_column *output) {

	GDF_REQUIRE(input->size == output->size, GDF_COLUMN_SIZE_MISMATCH);
	GDF_REQUIRE(output->dtype == GDF_INT16, GDF_UNSUPPORTED_DTYPE);

	hipStream_t stream;
	hipStreamCreate(&stream);

	gdf_size_type num_chars_bitmask = ( ( input->size +( GDF_VALID_BITSIZE - 1)) / GDF_VALID_BITSIZE );
	thrust::copy(thrust::cuda::par.on(stream), input->valid, input->valid + num_chars_bitmask, output->valid); // copy over valid bitmask

	if ( input->dtype == GDF_DATE64 ) {
		thrust::device_ptr<int64_t> input_ptr((int64_t *) input->data);
		thrust::device_ptr<int16_t> output_ptr((int16_t *) output->data);
		gdf_extract_datetime_day_date64_op op;
		thrust::transform(thrust::cuda::par.on(stream), thrust::detail::make_normal_iterator(input_ptr),
				thrust::detail::make_normal_iterator(input_ptr) + input->size, thrust::detail::make_normal_iterator(output_ptr), op);

	}else if (input->dtype == GDF_DATE32) {
		thrust::device_ptr<int32_t> input_ptr((int32_t *) input->data);
		thrust::device_ptr<int16_t> output_ptr((int16_t *) output->data);
		gdf_extract_datetime_day_date32_op op;
		thrust::transform(thrust::cuda::par.on(stream), thrust::detail::make_normal_iterator(input_ptr),
				thrust::detail::make_normal_iterator(input_ptr) + input->size, thrust::detail::make_normal_iterator(output_ptr), op);
	} else {
		return GDF_UNSUPPORTED_DTYPE;
	}

	hipStreamSynchronize(stream);
	hipStreamDestroy(stream);

	return GDF_SUCCESS;
}

gdf_error gdf_extract_datetime_hour(gdf_column *input, gdf_column *output) {

	GDF_REQUIRE(input->size == output->size, GDF_COLUMN_SIZE_MISMATCH);
	GDF_REQUIRE(output->dtype == GDF_INT16, GDF_UNSUPPORTED_DTYPE);
	GDF_REQUIRE(input->dtype != GDF_DATE32, GDF_UNSUPPORTED_DTYPE);

	hipStream_t stream;
	hipStreamCreate(&stream);

	gdf_size_type num_chars_bitmask = ( ( input->size +( GDF_VALID_BITSIZE - 1)) / GDF_VALID_BITSIZE );
	thrust::copy(thrust::cuda::par.on(stream), input->valid, input->valid + num_chars_bitmask, output->valid); // copy over valid bitmask

	if ( input->dtype == GDF_DATE64 ) {
		thrust::device_ptr<int64_t> input_ptr((int64_t *) input->data);
		thrust::device_ptr<int16_t> output_ptr((int16_t *) output->data);
		gdf_extract_datetime_hour_date64_op op;
		thrust::transform(thrust::cuda::par.on(stream), thrust::detail::make_normal_iterator(input_ptr),
				thrust::detail::make_normal_iterator(input_ptr) + input->size, thrust::detail::make_normal_iterator(output_ptr), op);

	} else {
		return GDF_UNSUPPORTED_DTYPE;
	}

	hipStreamSynchronize(stream);
	hipStreamDestroy(stream);

	return GDF_SUCCESS;
}

gdf_error gdf_extract_datetime_minute(gdf_column *input, gdf_column *output) {

	GDF_REQUIRE(input->size == output->size, GDF_COLUMN_SIZE_MISMATCH);
	GDF_REQUIRE(output->dtype == GDF_INT16, GDF_UNSUPPORTED_DTYPE);
	GDF_REQUIRE(input->dtype != GDF_DATE32, GDF_UNSUPPORTED_DTYPE);

	hipStream_t stream;
	hipStreamCreate(&stream);

	gdf_size_type num_chars_bitmask = ( ( input->size +( GDF_VALID_BITSIZE - 1)) / GDF_VALID_BITSIZE );
	thrust::copy(thrust::cuda::par.on(stream), input->valid, input->valid + num_chars_bitmask, output->valid); // copy over valid bitmask

	if ( input->dtype == GDF_DATE64 ) {
		thrust::device_ptr<int64_t> input_ptr((int64_t *) input->data);
		thrust::device_ptr<int16_t> output_ptr((int16_t *) output->data);
		gdf_extract_datetime_minute_date64_op op;
		thrust::transform(thrust::cuda::par.on(stream), thrust::detail::make_normal_iterator(input_ptr),
				thrust::detail::make_normal_iterator(input_ptr) + input->size, thrust::detail::make_normal_iterator(output_ptr), op);

	} else {
		return GDF_UNSUPPORTED_DTYPE;
	}

	hipStreamSynchronize(stream);
	hipStreamDestroy(stream);

	return GDF_SUCCESS;
}

gdf_error gdf_extract_datetime_second(gdf_column *input, gdf_column *output) {

	GDF_REQUIRE(input->size == output->size, GDF_COLUMN_SIZE_MISMATCH);
	GDF_REQUIRE(output->dtype == GDF_INT16, GDF_UNSUPPORTED_DTYPE);
	GDF_REQUIRE(input->dtype != GDF_DATE32, GDF_UNSUPPORTED_DTYPE);

	hipStream_t stream;
	hipStreamCreate(&stream);

	gdf_size_type num_chars_bitmask = ( ( input->size +( GDF_VALID_BITSIZE - 1)) / GDF_VALID_BITSIZE );
	thrust::copy(thrust::cuda::par.on(stream), input->valid, input->valid + num_chars_bitmask, output->valid); // copy over valid bitmask

	if ( input->dtype == GDF_DATE64 ) {
		thrust::device_ptr<int64_t> input_ptr((int64_t *) input->data);
		thrust::device_ptr<int16_t> output_ptr((int16_t *) output->data);
		gdf_extract_datetime_second_date64_op op;
		thrust::transform(thrust::cuda::par.on(stream), thrust::detail::make_normal_iterator(input_ptr),
				thrust::detail::make_normal_iterator(input_ptr) + input->size, thrust::detail::make_normal_iterator(output_ptr), op);

	} else {
		return GDF_UNSUPPORTED_DTYPE;
	}

	hipStreamSynchronize(stream);
	hipStreamDestroy(stream);

	return GDF_SUCCESS;
}



