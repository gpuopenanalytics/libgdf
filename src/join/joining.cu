#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2017, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <gdf/gdf.h>
#include <gdf/errorutils.h>
#include <limits>
#include <set>
#include <vector>

#include "joining.h"
#include "../gdf_table.cuh"

using namespace mgpu;

template <typename T>
void dump_mem(const char name[], const mem_t<T> & mem) {

    auto data = from_mem(mem);
    std::cout << name << " = " ;
    for (int i=0; i < data.size(); ++i) {
        std::cout << data[i] << ", ";
    }
    std::cout << "\n";
}


// Size limit due to use of int32 as join output.
// FIXME: upgrade to 64-bit
using output_index_type = int;
constexpr output_index_type MAX_JOIN_SIZE{std::numeric_limits<output_index_type>::max()};

// TODO This macro stuff will go away once Outer join is implemented
#define DEF_JOIN(Fn, T, Joiner)                                             \
gdf_error gdf_##Fn(gdf_column *leftcol, gdf_column *rightcol,               \
                   gdf_column *left_result, gdf_column *right_result) {     \
    using namespace mgpu;                                                   \
    if ( leftcol->dtype != rightcol->dtype) return GDF_UNSUPPORTED_DTYPE;   \
    if ( leftcol->size >= MAX_JOIN_SIZE ) return GDF_COLUMN_SIZE_TOO_BIG;   \
    if ( rightcol->size >= MAX_JOIN_SIZE ) return GDF_COLUMN_SIZE_TOO_BIG;  \
    standard_context_t context;                                             \
    auto output = Joiner((T*)leftcol->data, leftcol->size,                  \
                                (T*)rightcol->data, rightcol->size,         \
                                less_t<T>(), context);                      \
    *left_result = output.first;                                            \
    *right_result = output.second;                                          \
    CUDA_CHECK_LAST();                                                      \
    return GDF_SUCCESS;                                                     \
}

#define DEF_JOIN_GENERIC(Fn)                                                            \
gdf_error gdf_##Fn##_generic(gdf_column *leftcol, gdf_column * rightcol,                \
                             gdf_column *l_result, gdf_column *r_result) {              \
    switch ( leftcol->dtype ){                                                          \
    case GDF_INT8:      return gdf_##Fn##_i8 (leftcol, rightcol, l_result, r_result);   \
    case GDF_INT16:     return gdf_##Fn##_i16(leftcol, rightcol, l_result, r_result);   \
    case GDF_INT32:     return gdf_##Fn##_i32(leftcol, rightcol, l_result, r_result);   \
    case GDF_INT64:     return gdf_##Fn##_i64(leftcol, rightcol, l_result, r_result);   \
    case GDF_FLOAT32:   return gdf_##Fn##_f32(leftcol, rightcol, l_result, r_result);   \
    case GDF_FLOAT64:   return gdf_##Fn##_f64(leftcol, rightcol, l_result, r_result);   \
    case GDF_DATE32:    return gdf_##Fn##_i32(leftcol, rightcol, l_result, r_result);   \
    case GDF_DATE64:    return gdf_##Fn##_i64(leftcol, rightcol, l_result, r_result);   \
    case GDF_TIMESTAMP: return gdf_##Fn##_i64(leftcol, rightcol, l_result, r_result);   \
    default: return GDF_UNSUPPORTED_DTYPE;                                              \
    }                                                                                   \
}

#define DEF_OUTER_JOIN(Fn, T) DEF_JOIN(outer_join_ ## Fn, T, outer_join)
DEF_JOIN_GENERIC(outer_join)
DEF_OUTER_JOIN(i8,  int8_t)
DEF_OUTER_JOIN(i16, int16_t)
DEF_OUTER_JOIN(i32, int32_t)
DEF_OUTER_JOIN(i64, int64_t)
DEF_OUTER_JOIN(f32, int32_t)
DEF_OUTER_JOIN(f64, int64_t)

/* --------------------------------------------------------------------------*/
/** 
 * @Synopsis Computes the Join result between two tables using the hash-based implementation. 
 * 
 * @Param num_cols The number of columns to join
 * @Param leftcol The left set of columns to join
 * @Param rightcol The right set of columns to join
 * @Param out_result The result of the join operation. The first n/2 elements of the
   output are the left indices, the last n/2 elements of the output are the right indices.
   @tparam join_type The type of join to be performed
 * 
 * @Returns Upon successful computation, returns GDF_SUCCESS. Otherwise returns appropriate error code 
 */
/* ----------------------------------------------------------------------------*/
template <JoinType join_type, 
          typename size_type>
gdf_error hash_join(size_type num_cols, gdf_column **leftcol, gdf_column **rightcol,
                    gdf_column *l_result, gdf_column *r_result)
{
  // Wrap the set of gdf_columns in a gdf_table class
  std::unique_ptr< gdf_table<size_type> > left_table(new gdf_table<size_type>(num_cols, leftcol));
  std::unique_ptr< gdf_table<size_type> > right_table(new gdf_table<size_type>(num_cols, rightcol));

  return join_hash<join_type, output_index_type>(*left_table, 
                                                        *right_table, 
                                                        l_result, 
                                                        r_result);
}

template <JoinType join_type>
struct SortJoin {
template<typename launch_arg_t = mgpu::empty_t,
  typename a_it, typename b_it, typename comp_t>
    std::pair<gdf_column, gdf_column>
    operator()(a_it a, int a_count, b_it b, int b_count,
               comp_t comp, context_t& context) {
        return std::pair<gdf_column, gdf_column>();
    }
};

template <>
struct SortJoin<JoinType::INNER_JOIN> {
template<typename launch_arg_t = mgpu::empty_t,
  typename a_it, typename b_it, typename comp_t>
    std::pair<gdf_column, gdf_column>
    operator()(a_it a, int a_count, b_it b, int b_count,
               comp_t comp, context_t& context) {
        return inner_join(a, a_count, b, b_count, comp, context);
    }
};

template <>
struct SortJoin<JoinType::LEFT_JOIN> {
  template<typename launch_arg_t = mgpu::empty_t,
    typename a_it, typename b_it, typename comp_t>
    std::pair<gdf_column, gdf_column>
    operator()(a_it a, int a_count, b_it b, int b_count,
               comp_t comp, context_t& context) {
        return left_join(a, a_count, b, b_count, comp, context);
      }
};

template <JoinType join_type, typename T>
gdf_error sort_join_typed(gdf_column *leftcol, gdf_column *rightcol,
                          gdf_column *left_result, gdf_column *right_result,
                          gdf_context *ctxt) 
{
  using namespace mgpu;
  gdf_error err = GDF_SUCCESS;

  standard_context_t context(false);
  SortJoin<join_type> sort_based_join;
  auto output = sort_based_join(static_cast<T*>(leftcol->data), leftcol->size,
                                       static_cast<T*>(rightcol->data), rightcol->size,
                                       less_t<T>(), context);
  *left_result = output.first;
  *right_result = output.second;
  CUDA_CHECK_LAST();

  return err;
}

/* --------------------------------------------------------------------------*/
/** 
 * @Synopsis  Computes the join operation between a single left and single right column
 using the sort based implementation.
 * 
 * @Param leftcol The left column to join
 * @Param rightcol The right column to join
 * @Param out_result The output of the join operation
 * @Param ctxt Structure that determines various run parameters, such as if the inputs
 are already sorted.
   @tparama join_type The type of join to perform
 * 
 * @Returns GDF_SUCCESS upon succesful completion of the join, otherwise returns 
 appropriate error code.
 */
/* ----------------------------------------------------------------------------*/
template <JoinType join_type>
gdf_error sort_join(gdf_column *leftcol, gdf_column *rightcol,
                    gdf_column *l_result, gdf_column *r_result,
                    gdf_context *ctxt)
{

  if(GDF_SORT != ctxt->flag_method) return GDF_INVALID_API_CALL;

  switch ( leftcol->dtype ){
    case GDF_INT8:      return sort_join_typed<join_type, int8_t>(leftcol, rightcol, l_result, r_result, ctxt);
    case GDF_INT16:     return sort_join_typed<join_type,int16_t>(leftcol, rightcol, l_result, r_result, ctxt);
    case GDF_INT32:     return sort_join_typed<join_type,int32_t>(leftcol, rightcol, l_result, r_result, ctxt);
    case GDF_INT64:     return sort_join_typed<join_type,int64_t>(leftcol, rightcol, l_result, r_result, ctxt);
    case GDF_FLOAT32:   return sort_join_typed<join_type,int32_t>(leftcol, rightcol, l_result, r_result, ctxt);
    case GDF_FLOAT64:   return sort_join_typed<join_type,int64_t>(leftcol, rightcol, l_result, r_result, ctxt);
    case GDF_DATE32:    return sort_join_typed<join_type,int32_t>(leftcol, rightcol, l_result, r_result, ctxt);
    case GDF_DATE64:    return sort_join_typed<join_type,int64_t>(leftcol, rightcol, l_result, r_result, ctxt);
    case GDF_TIMESTAMP: return sort_join_typed<join_type,int64_t>(leftcol, rightcol, l_result, r_result, ctxt);
    default: return GDF_UNSUPPORTED_DTYPE;
  }
}

template
gdf_error sort_join<JoinType::INNER_JOIN>(gdf_column *leftcol, gdf_column *rightcol,
                                          gdf_column *l_result, gdf_column *r_result,
                                          gdf_context *ctxt);
template
gdf_error sort_join<JoinType::LEFT_JOIN>(gdf_column *leftcol, gdf_column *rightcol,
                                         gdf_column *l_result, gdf_column *r_result,
                                         gdf_context *ctxt);

/* --------------------------------------------------------------------------*/
/** 
 * @Synopsis  Computes the join operation between two sets of columns
 * 
 * @Param num_cols The number of columns to join
 * @Param leftcol The left set of columns to join
 * @Param rightcol The right set of columns to join
 * @Param out_result The result of the join operation. The output is structured such that
 * the pair (i, i + output_size/2) is the (left, right) index of matching rows.
 * @Param join_context A structure that determines various run parameters, such as
   whether to perform a hash or sort based join
 * @tparam join_type The type of join to be performed
 * 
 * @Returns GDF_SUCCESS upon succesfull compute, otherwise returns appropriate error code
 */
/* ----------------------------------------------------------------------------*/
template <JoinType join_type>
gdf_error join_call( int num_cols, gdf_column **leftcol, gdf_column **rightcol,
                     gdf_column *left_result, gdf_column *right_result,
                     gdf_context *join_context)
{

  if( (0 == num_cols) || (nullptr == leftcol) || (nullptr == rightcol))
    return GDF_DATASET_EMPTY;

  if(nullptr == join_context)
    return GDF_INVALID_API_CALL;

  const auto left_col_size = leftcol[0]->size;
  const auto right_col_size = rightcol[0]->size;
  
  // Check that the number of rows does not exceed the maximum
  if(left_col_size >= MAX_JOIN_SIZE) return GDF_COLUMN_SIZE_TOO_BIG;
  if(right_col_size >= MAX_JOIN_SIZE) return GDF_COLUMN_SIZE_TOO_BIG;

  // If both frames are empty, return immediately
  if((0 == left_col_size ) && (0 == right_col_size)) {
    return GDF_SUCCESS;
  }

  // If left join and the left table is empty, return immediately
  if( (JoinType::LEFT_JOIN == join_type) && (0 == left_col_size)){
    return GDF_SUCCESS;
  }

  // If Inner Join and either table is empty, return immediately
  if( (JoinType::INNER_JOIN == join_type) && 
      ((0 == left_col_size) || (0 == right_col_size)) ){
    return GDF_SUCCESS;
  }

  // check that the columns data are not null, have matching types, 
  // and the same number of rows
  for (int i = 0; i < num_cols; i++) {
    if((right_col_size > 0) && (nullptr == rightcol[i]->data)){
     return GDF_DATASET_EMPTY;
    } 
    if((left_col_size > 0) && (nullptr == leftcol[i]->data)){
     return GDF_DATASET_EMPTY;
    } 
    if(rightcol[i]->dtype != leftcol[i]->dtype) return GDF_JOIN_DTYPE_MISMATCH;
    if(left_col_size != leftcol[i]->size) return GDF_COLUMN_SIZE_MISMATCH;
    if(right_col_size != rightcol[i]->size) return GDF_COLUMN_SIZE_MISMATCH;
  }

  gdf_method join_method = join_context->flag_method; 

  switch(join_method)
  {
    case GDF_HASH:
      {
        return hash_join<join_type, int64_t>(num_cols, leftcol, rightcol, left_result, right_result);
      }
    case GDF_SORT:
      {
        // Sort based joins only support single column joins
        if(1 == num_cols)
        {
          return sort_join<join_type>(leftcol[0], rightcol[0], left_result, right_result, join_context);
        }
        else
        {
          return GDF_JOIN_TOO_MANY_COLUMNS;
        }
      }
    default:
      return GDF_UNSUPPORTED_METHOD;
  }

}

template <JoinType join_type, typename size_type, typename index_type>
gdf_error construct_join_output_df(
        std::vector<gdf_column*>& ljoincol,
        std::vector<gdf_column*>& rjoincol,
        gdf_column **left_cols, 
        int num_left_cols,
        int left_join_cols[],
        gdf_column **right_cols,
        int num_right_cols,
        int right_join_cols[],
        int num_cols_to_join,
        int result_num_cols,
        gdf_column ** result_cols,
        gdf_column * left_indices,
        gdf_column * right_indices) {
    //create left and right input table with columns not joined on
    std::vector<gdf_column*> lnonjoincol;
    std::vector<gdf_column*> rnonjoincol;
    std::set<int> l_join_indices, r_join_indices;
    for (int i = 0; i < num_cols_to_join; ++i) {
        l_join_indices.insert(left_join_cols[i]);
        r_join_indices.insert(right_join_cols[i]);
    }
    for (int i = 0; i < num_left_cols; ++i) {
        if (l_join_indices.find(i) == l_join_indices.end()) {
            lnonjoincol.push_back(left_cols[i]);
        }
    }
    for (int i = 0; i < num_right_cols; ++i) {
        if (r_join_indices.find(i) == r_join_indices.end()) {
            rnonjoincol.push_back(right_cols[i]);
        }
    }
    //TODO : Invalid api

    size_t join_size = left_indices->size;
    int left_table_end = num_left_cols - num_cols_to_join;
    int right_table_begin = num_left_cols;

    //create left and right output column data buffers
    for (int i = 0; i < left_table_end; ++i) {
        gdf_column_view(result_cols[i], nullptr, nullptr, join_size, lnonjoincol[i]->dtype);
        int col_width; get_column_byte_width(result_cols[i], &col_width);
        CUDA_TRY( hipMalloc(&(result_cols[i]->data), col_width * join_size) );
        CUDA_TRY( hipMalloc(&(result_cols[i]->valid), sizeof(gdf_valid_type)*gdf_get_num_chars_bitmask(join_size)) );
        CUDA_TRY( hipMemset(result_cols[i]->valid, 0, sizeof(gdf_valid_type)*gdf_get_num_chars_bitmask(join_size)) );
    }
    for (int i = right_table_begin; i < result_num_cols; ++i) {
        gdf_column_view(result_cols[i], nullptr, nullptr, join_size, rnonjoincol[i - right_table_begin]->dtype);
        int col_width; get_column_byte_width(result_cols[i], &col_width);
        CUDA_TRY( hipMalloc(&(result_cols[i]->data), col_width * join_size) );
        CUDA_TRY( hipMalloc(&(result_cols[i]->valid), sizeof(gdf_valid_type)*gdf_get_num_chars_bitmask(join_size)) );
        CUDA_TRY( hipMemset(result_cols[i]->valid, 0, sizeof(gdf_valid_type)*gdf_get_num_chars_bitmask(join_size)) );
    }
    //create joined output column data buffers
    for (int join_index = 0; join_index < num_cols_to_join; ++join_index) {
        int i = left_table_end + join_index;
        gdf_column_view(result_cols[i], nullptr, nullptr, join_size, left_cols[left_join_cols[join_index]]->dtype);
        int col_width; get_column_byte_width(result_cols[i], &col_width);
        CUDA_TRY( hipMalloc(&(result_cols[i]->data), col_width * join_size) );
        CUDA_TRY( hipMalloc(&(result_cols[i]->valid), sizeof(gdf_valid_type)*gdf_get_num_chars_bitmask(join_size)) );
        CUDA_TRY( hipMemset(result_cols[i]->valid, 0, sizeof(gdf_valid_type)*gdf_get_num_chars_bitmask(join_size)) );
    }

    gdf_table<size_type> l_i_table(lnonjoincol.size(), lnonjoincol.data());
    gdf_table<size_type> r_i_table(rnonjoincol.size(), rnonjoincol.data());
    gdf_table<size_type> j_i_table(ljoincol.size(), ljoincol.data());

    gdf_table<size_type> l_table(num_left_cols - num_cols_to_join, result_cols);
    gdf_table<size_type> r_table(num_right_cols - num_cols_to_join, result_cols + right_table_begin);
    gdf_table<size_type> j_table(num_cols_to_join, result_cols + left_table_end);

    gdf_error err{GDF_SUCCESS};
    err = l_i_table.gather(static_cast<index_type*>(left_indices->data),
            l_table, join_type != JoinType::INNER_JOIN);
    if (err != GDF_SUCCESS) { return err; }
    err = r_i_table.gather(static_cast<index_type*>(right_indices->data),
            r_table, join_type != JoinType::INNER_JOIN);
    if (err != GDF_SUCCESS) { return err; }
    err = j_i_table.gather(static_cast<index_type*>(left_indices->data),
            j_table, join_type != JoinType::INNER_JOIN);
    return err;
}

template <JoinType join_type, typename size_type, typename index_type>
gdf_error join_call_compute_df(
                         gdf_column **left_cols, 
                         int num_left_cols,
                         int left_join_cols[],
                         gdf_column **right_cols,
                         int num_right_cols,
                         int right_join_cols[],
                         int num_cols_to_join,
                         int result_num_cols,
                         gdf_column **result_cols,
                         gdf_column * left_indices,
                         gdf_column * right_indices,
                         gdf_context *join_context) {
    //return error if the inputs are invalid
    if ((left_cols == nullptr)  ||
        (right_cols == nullptr)) { return GDF_DATASET_EMPTY; }

    //check if combined join output is expected
    bool compute_df = (result_cols != nullptr);

    //return error if no output pointers are valid
    if ( ((left_indices == nullptr)||(right_indices == nullptr)) &&
         (!compute_df) ) { return GDF_DATASET_EMPTY; }

    //If index outputs are not requested, create columns to store them
    //for computing combined join output
    gdf_column * left_index_out = left_indices;
    gdf_column * right_index_out = right_indices;

    using gdf_col_pointer = typename std::unique_ptr<gdf_column, std::function<void(gdf_column*)>>;
    auto gdf_col_deleter = [](gdf_column* col){
        col->size = 0;
        if (col->data)  { hipFree(col->data);  }
        if (col->valid) { hipFree(col->valid); }
    };
    gdf_col_pointer l_index_temp, r_index_temp;

    if (nullptr == left_indices) {
        l_index_temp = {new gdf_column, gdf_col_deleter};
        left_index_out = l_index_temp.get();
    }

    if (nullptr == right_indices) {
        r_index_temp = {new gdf_column, gdf_col_deleter};
        right_index_out = r_index_temp.get();
    }

    //get column pointers to join on
    std::vector<gdf_column*> ljoincol;
    std::vector<gdf_column*> rjoincol;
    for (int i = 0; i < num_cols_to_join; ++i) {
        ljoincol.push_back(left_cols[ left_join_cols[i] ]);
        rjoincol.push_back(right_cols[ right_join_cols[i] ]);
    }


    gdf_error join_err = join_call<join_type>(num_cols_to_join,
            ljoincol.data(), rjoincol.data(),
            left_index_out, right_index_out,
            join_context);
    //If compute_df is false then left_index_out or right_index_out
    //was not dynamically allocated.
    if ((!compute_df) || (GDF_SUCCESS != join_err)) {
        return join_err;
    }

    gdf_error df_err =
        construct_join_output_df<join_type, size_type, index_type>(
            ljoincol, rjoincol,
            left_cols, num_left_cols, left_join_cols,
            right_cols, num_right_cols, right_join_cols,
            num_cols_to_join, result_num_cols, result_cols,
            left_index_out, right_index_out);

    l_index_temp.reset(nullptr);
    r_index_temp.reset(nullptr);

    CUDA_CHECK_LAST();

    return df_err;
}

gdf_error gdf_left_join(
                         gdf_column **left_cols, 
                         int num_left_cols,
                         int left_join_cols[],
                         gdf_column **right_cols,
                         int num_right_cols,
                         int right_join_cols[],
                         int num_cols_to_join,
                         int result_num_cols,
                         gdf_column **result_cols,
                         gdf_column * left_indices,
                         gdf_column * right_indices,
                         gdf_context *join_context) {
    return join_call_compute_df<JoinType::LEFT_JOIN, int64_t, output_index_type>(
                     left_cols, 
                     num_left_cols,
                     left_join_cols,
                     right_cols,
                     num_right_cols,
                     right_join_cols,
                     num_cols_to_join,
                     result_num_cols,
                     result_cols,
                     left_indices,
                     right_indices,
                     join_context);
}

gdf_error gdf_inner_join(
                         gdf_column **left_cols, 
                         int num_left_cols,
                         int left_join_cols[],
                         gdf_column **right_cols,
                         int num_right_cols,
                         int right_join_cols[],
                         int num_cols_to_join,
                         int result_num_cols,
                         gdf_column **result_cols,
                         gdf_column * left_indices,
                         gdf_column * right_indices,
                         gdf_context *join_context) {
    return join_call_compute_df<JoinType::INNER_JOIN, int64_t, output_index_type>(
                     left_cols, 
                     num_left_cols,
                     left_join_cols,
                     right_cols,
                     num_right_cols,
                     right_join_cols,
                     num_cols_to_join,
                     result_num_cols,
                     result_cols,
                     left_indices,
                     right_indices,
                     join_context);
}
