#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

//Quantile (percentile) testing


#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include "thrust_rmm_allocator.h"

#include <iostream>
#include <vector>
#include <string>

#include <cassert>
#include <cmath>

#include <gdf/gdf.h>
#include <gdf/utils.h>
#include <gdf/errorutils.h>
#include <gdf/cffi/functions.h>

#include "gtest/gtest.h"

#include "gdf_test_fixtures.h"

#include "quantiles.hpp"

// Vector set to use rmmAlloc and rmmFree.
template <typename T>
using Vector = thrust::device_vector<T, rmm_allocator<T>>;


template<typename T, typename Allocator, template<typename, typename> class Vector>
__host__ __device__
void print_v(const Vector<T, Allocator>& v, std::ostream& os)
{
  thrust::copy(v.begin(), v.end(), std::ostream_iterator<T>(os,","));
  os<<"\n";
}


template<typename VType>
void f_quantile_tester(gdf_column* col_in, std::vector<VType>& v_out_exact, std::vector<std::vector<double>>& v_out_m)
{
  std::vector<std::string> methods{"lin_interp", "lower", "higher", "midpoint", "nearest"};
  size_t n_methods = methods.size();
  
  std::vector<double> qvals{0.0, 0.25, 0.33, 0.5, 1.0};
  size_t n_qs = qvals.size();
  
  assert( n_methods == methods.size() );
  gdf_context ctxt{0, static_cast<gdf_method>(0), 0, 1};
  
  for(size_t j = 0; j<n_qs; ++j)
    {
      VType res = 0;
      auto q = qvals[j];
      gdf_error ret = gdf_quantile_aprrox(col_in, q, &res, &ctxt);
      v_out_exact[j] = res;
      EXPECT_EQ( ret, GDF_SUCCESS) << "approx " << " returns unexpected failure\n";
      
      for(size_t i = 0;i<n_methods;++i)
        {
          double rt = 0;
          ret = gdf_quantile_exact(col_in, static_cast<gdf_quantile_method>(i), q, &rt, &ctxt);
          v_out_m[j][i] = rt;
          
          EXPECT_EQ( ret, GDF_SUCCESS) << "exact " << methods[i] << " returns unexpected failure\n";
        }
    }
}

struct gdf_quantile : public GdfTest {};

TEST_F(gdf_quantile, DoubleVector)
{
  using VType = double;
  std::vector<VType> v{6.8, 0.15, 3.4, 4.17, 2.13, 1.11, -1.01, 0.8, 5.7};
  Vector<VType> d_in = v;
  
  gdf_column col_in;
  col_in.size = d_in.size();
  col_in.data = d_in.data().get();
  col_in.valid = nullptr;
  col_in.dtype = GDF_FLOAT64;

  size_t n_qs = 5;
  size_t n_methods = 5;

  std::vector<VType> v_baseline_approx{-1.01, 0.15, 0.15, 1.11, 6.8};
  std::vector<std::vector<double>> v_baseline_exact{
    {-1.01, -1.01, 0.15, -0.43, -1.01},
      {0.3125, 0.15, 0.8, 0.475, 0.15},
        {0.7805, 0.15, 0.8, 0.475, 0.8},
          {1.62, 1.11, 2.13, 1.62, 2.13},
            {6.8, 6.8, 6.8, 6.8, 6.8}};
  
  std::vector<VType> v_out_approx(n_qs, 0);
  std::vector<std::vector<double>> v_out_exact(n_qs, std::vector<double>(n_methods,0.0));

  f_quantile_tester<VType>(&col_in, v_out_approx, v_out_exact);

  for(size_t i=0; i<n_qs;++i)
    {
      double delta = std::abs(static_cast<double>(v_baseline_approx[i] - v_out_approx[i]));
      bool flag = delta < 1.0e-8;
      EXPECT_EQ( flag, true ) << i <<"-th quantile deviates from baseline by: " << delta;
    }

  for(size_t i=0; i<n_qs;++i)
    {
      for(size_t j=0; j < n_methods; ++j)
        {
          double delta = std::abs(static_cast<double>(v_baseline_exact[i][j] - v_out_exact[i][j]));
          bool flag = delta < 1.0e-8;
          EXPECT_EQ( flag, true ) << i <<"-th quantile on " << j << "-th deviates from baseline by: " << delta;
        }
    }
}

TEST_F(gdf_quantile, IntegerVector)
{
  using VType = int32_t;
  std::vector<VType> v{7, 0, 3, 4, 2, 1, -1, 1, 6};;
  Vector<VType> d_in = v;
  
  gdf_column col_in;
  col_in.size = d_in.size();
  col_in.data = d_in.data().get();
  col_in.valid = nullptr;
  col_in.dtype = GDF_INT32;

  size_t n_qs = 5;
  size_t n_methods = 5;

  std::vector<VType> v_baseline_approx{-1, 0, 0, 1, 7};
  std::vector<std::vector<double>> v_baseline_exact{
    {-1, -1, 0, -0.5, -1},
      {0.25, 0, 1, 0.5, 0},
        {0.97, 0, 1, 0.5, 1},
          {1.5, 1, 2, 1.5, 2},
            {7, 7, 7, 7, 7}};
  
  std::vector<VType> v_out_approx(n_qs, 0);
  std::vector<std::vector<double>> v_out_exact(n_qs, std::vector<double>(n_methods,0.0));

  f_quantile_tester<VType>(&col_in, v_out_approx, v_out_exact);

  for(size_t i=0; i<n_qs;++i)
    {
      double delta = std::abs(static_cast<double>(v_baseline_approx[i] - v_out_approx[i]));
      bool flag = delta < 1.0e-8;
      EXPECT_EQ( flag, true ) << i <<"-th quantile deviates from baseline by: " << delta;
    }

  for(size_t i=0; i<n_qs;++i)
    {
      for(size_t j=0; j < n_methods; ++j)
        {
          double delta = std::abs(static_cast<double>(v_baseline_exact[i][j] - v_out_exact[i][j]));
          bool flag = delta < 1.0e-8;
          EXPECT_EQ( flag, true ) << i <<"-th quantile on " << j << "-th deviates from baseline by: " << delta;
        }
    }
}


int main(int argc, char **argv) {
  ::testing::InitGoogleTest(&argc, argv);
  return RUN_ALL_TESTS();
}


